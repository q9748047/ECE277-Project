#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel_madd(int* A, int* B, int* C, int M, int N);

void cu_madd(int* A, int* B, int* C, int M, int N)
{
	int *d_a, *d_b, *d_c;

	dim3 blk;
	blk.x = 16; blk.y = 16;

	dim3 grid;
	grid.x = (M + blk.x - 1) / blk.x;
	grid.y = (N + blk.y - 1) / blk.y;
	grid.z = 1;

	int size = sizeof(unsigned int)*M*N;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);

	kernel_madd << < grid, blk >> > (d_a, d_b, d_c, M, N);

	hipMemcpy(C, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

__global__ void kernel_madd(int* A, int* B, int* C, int M, int N)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int idx = iy * M + ix;

	if (idx == 0)
		printf("cuda matrix (%d, %d) addition\n", N, M);

	if (ix < M && iy < N)
		C[idx] = A[idx] + B[idx];
}

